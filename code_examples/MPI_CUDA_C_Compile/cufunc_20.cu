#include "hip/hip_runtime.h"
extern "C" {
	#include "20.h"
}

__global__ void cusum( int rank, float *data ) {
	int i = threadIdx.x;
	data[i] += rank;
}


__host__ void cufunc( int rank, float *data ) {
	float *dev_data;
	int array_size = CHUNKSIZE*sizeof(float);

	hipMalloc ( (void**) &dev_data, array_size );

	hipMemcpy( dev_data, data, array_size, hipMemcpyHostToDevice );
	cusum <<<dim3(1),dim3(CHUNKSIZE)>>> ( rank, dev_data );
	hipMemcpy( data, dev_data, array_size, hipMemcpyDeviceToHost );
}

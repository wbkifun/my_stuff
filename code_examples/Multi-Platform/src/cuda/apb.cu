#include "hip/hip_runtime.h"
#include "param1.h"
#include "apb_ext.h"


extern "C" {


__global__ void apb(int shift_gid, int nx, double *a, double *b, double *c) {
	int gid = blockDim.x * blockIdx.x + threadIdx.x + shift_gid;

	if (gid >= nx) return;

	c[gid] = KK*a[gid] + bpc(b[gid], c[gid]);
}


}	// extern "C"

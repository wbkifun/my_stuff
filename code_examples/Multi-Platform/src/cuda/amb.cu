#include "hip/hip_runtime.h"
#include "param1.h"
#include "amb_ext1.h"


extern "C" {


__global__ void amb(int shift_gid, int nx, double *a, double *b, double *c) {
	int gid = blockDim.x * blockIdx.x + threadIdx.x + shift_gid;
	
	if (gid >= nx) return;

	bmc(LLL, b, c);
	c[gid] = KK*a[gid] + c[gid];
}


}	// extern "C"

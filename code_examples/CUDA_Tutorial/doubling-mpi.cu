#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>


__global__ void doubling(int n, float *a) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int i;

	while(tid < n) {
		a[tid] *= 2;
		for(i=0; i<1000; i++) a[tid] *= 1;
		tid += blockDim.x * gridDim.x;
	}
}


int main(int argc, char **argv) {
	int count, rank, size;
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	hipGetDeviceCount(&count);
	if( rank==0 ) {
		printf("Number of CUDA-enabled GPU devices: %d\n", count);
		printf("MPI size: %d\n", size);
	}

	int i;
	int n=65535*3500;

	float *a, *a_dev;
	a = (float *)malloc(n*sizeof(float));
	hipMalloc((void**)&a_dev, n*sizeof(float));

	for(i=0; i<n; i++) a[i] = 1.; 
	hipMemcpy(a_dev, a, n*sizeof(float), hipMemcpyHostToDevice);

	doubling<<<65535, 256>>>(n, a_dev);
	hipMemcpy(a, a_dev, n*sizeof(float), hipMemcpyDeviceToHost);

	for(i=0; i<n; i++) {
		if(abs(a[i] - 2.) > 1e-5) printf("a[%d] = %g\n", i, a[i]); 
	}

	free(a);
	hipFree(a_dev);
	MPI_Finalize();
	return 0;
}

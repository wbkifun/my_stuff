#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <sys/time.h>

void vecadd(int n, float *a, float *b, float *c) {
	for(int i=0; i<n; i++) {
		c[i] = a[i] + b[i];
	}
}

__global__ void vecadd_kernel(int n, float *a, float *b, float *c, int offset) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x + offset;

	if( tid<n ) {
		c[tid] = a[tid] + b[tid];
	}
}

void vecadd_gpu(int n, float *a, float *b, float *c) {
	int i;
	int tpb =256;	// thread per block
	int max_bpg = 65535;
	int ng = n/(max_bpg*tpb); 	// number of grid
	for(i=0; i<ng; i++) {
		vecadd_kernel<<<max_bpg,tpb>>>(n, a, b, c, i*max_bpg*tpb);
	}
	if( n%(max_bpg*tpb)!=0 ) {
		int nn = n-ng*max_bpg*tpb;
		int bpg = nn%tpb==0 ? nn/tpb : nn/tpb+1;
		vecadd_kernel<<<bpg,tpb>>>(n, a, b, c, ng*max_bpg*tpb);
	}
}


int main(int argc, char **argv) {
	int count, rank, size;
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &size);
	//MPI_Status status;

	hipGetDeviceCount(&count);
	if( rank==0 ) {
		printf("MPI size: %d\n", size);
		printf("Number of CUDA-enabled GPU devices: %d\n", count);
	}
	hipSetDevice(rank);

	int i;
	int tn=800000000;
	int n = tn/size;

	float *ta, *tb, *tc;
	if( rank==0 ) {
		ta = (float *)malloc(tn*sizeof(float));
		tb = (float *)malloc(tn*sizeof(float));
		tc = (float *)malloc(tn*sizeof(float));

		for(i=0; i<tn; i++) {
			ta[i] = rand()/(RAND_MAX+1.);
			tb[i] = rand()/(RAND_MAX+1.);
		}
	}

	float *a, *b;
	a = (float *)malloc(n*sizeof(float));
	b = (float *)malloc(n*sizeof(float));

	MPI_Scatter(ta, n, MPI_FLOAT, a, n, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Scatter(tb, n, MPI_FLOAT, b, n, MPI_FLOAT, 0, MPI_COMM_WORLD);

	float *a_dev, *b_dev, *c_dev;
	hipMalloc((void**)&a_dev, n*sizeof(float));
	hipMalloc((void**)&b_dev, n*sizeof(float));
	hipMalloc((void**)&c_dev, n*sizeof(float));

	hipMemcpy(a_dev, a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b, n*sizeof(float), hipMemcpyHostToDevice);

	if( rank==0 ) {
		hipEvent_t t0, t1;
		hipEventCreate(&t0);
		hipEventCreate(&t1);
		hipEventRecord(t0, 0);

		vecadd_gpu(n, a_dev, b_dev, c_dev);

		hipEventRecord(t1, 0);
		hipEventSynchronize(t1);
		float gpu_time;
		hipEventElapsedTime(&gpu_time, t0, t1);
		hipEventDestroy(t0);
		hipEventDestroy(t1);
		printf("GPU run-time: %1.3f sec\n", gpu_time*1e-3);

		struct timeval t2, t3;
		gettimeofday(&t2, NULL);
		vecadd(tn, ta, tb, tc);
		gettimeofday(&t3, NULL);
		float cpu_time=t3.tv_sec+t3.tv_usec*1e-6 - (t2.tv_sec+t2.tv_usec*1e-6);
		printf("CPU run-time: %1.3f sec\n", cpu_time);
	}
	else vecadd_gpu(n, a_dev, b_dev, c_dev);


	hipMemcpy(b, c_dev, n*sizeof(float), hipMemcpyDeviceToHost);
	MPI_Gather(b, n, MPI_FLOAT, tb, n, MPI_FLOAT, 0, MPI_COMM_WORLD);

	if( rank==0 ) {
		printf("tn=%d, n=%d\n", tn, n);
		printf("Check results..");
		float diff;
		for(i=0; i<tn; i++) {
			diff = fabs(tc[i]-tb[i]);
			if(diff > 1e-7) break;
		}
		if(diff > 1e-7) printf("Mismatch!\n");
		else printf("OK!\n");
	}

	MPI_Finalize();
	return 0;
}

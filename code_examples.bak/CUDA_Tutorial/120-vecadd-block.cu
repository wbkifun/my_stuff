
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void print_array(int n, char str, float *a) {
	printf("%c:  ", str);
	for(int i=0; i<n; i++) printf("\t%f", a[i]);
	printf("\n");
}

void vecadd(int n, float *a, float *b, float *c) {
	for(int i=0; i<n; i++) {
		c[i] = a[i] + b[i];
	}
}

__global__ void vecadd_gpu(int n, float *a, float *b, float *c, int offset) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x + offset;

	if( tid<n ) {
		c[tid] = a[tid] + b[tid];
	}
}


int main() {
	int i;
	int n=300000000;
	float *a, *b, *c;

	// allocation in host memory
	a = (float *)malloc(n*sizeof(float));
	b = (float *)malloc(n*sizeof(float));
	c = (float *)malloc(n*sizeof(float));

	// initialize
	for(i=0; i<n; i++) {
		//a[i] = i+5.5;
		//b[i] = -1.2*i;
		a[i] = rand()/(RAND_MAX+1.);
		b[i] = rand()/(RAND_MAX+1.);
	}
	//print_array(n, 'a', a);
	//print_array(n, 'b', b);

	// call the function
	vecadd(n, a, b, c);
	//printf("results from CPU\n");
	//print_array(n, 'c', c);

	// allocation in device memory
	float *a_dev, *b_dev, *c_dev;
	hipMalloc((void**)&a_dev, n*sizeof(float));
	hipMalloc((void**)&b_dev, n*sizeof(float));
	hipMalloc((void**)&c_dev, n*sizeof(float));
	
	// copy arrays 'a' and 'b' to the device
	hipMemcpy(a_dev, a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b, n*sizeof(float), hipMemcpyHostToDevice);
	
	// call the kernel
	int tpb =256;	// thread per block
    int max_bpg = 65535;
	int ng = n/(max_bpg*tpb); 	// number of grid
	for(i=0; i<ng; i++) {
		vecadd_gpu<<<max_bpg,tpb>>>(n, a_dev, b_dev, c_dev, i*max_bpg*tpb);
	}
	if( n%(max_bpg*tpb)!=0 ) {
		int nn = n-ng*max_bpg*tpb;
		int bpg = nn%tpb==0 ? nn/tpb : nn/tpb+1;
		vecadd_gpu<<<bpg,tpb>>>(n, a_dev, b_dev, c_dev, ng*max_bpg*tpb);
	}
			
	// copy array 'c' back from the device to the host
	float *c2;
	c2 = (float *)malloc(n*sizeof(float));
	hipMemcpy(c2, c_dev, n*sizeof(float), hipMemcpyDeviceToHost);
	//printf("results from GPU\n");
	//print_array(n, 'c', c2);

	printf("n=%d\n", n);
	printf("Check results..");
	float diff;
	for(i=0; i<n; i++) {
		diff = fabs(c2[i]-c[i]);
		if(diff > 1e-7) break;
	}
	if(diff > 1e-7) printf("Mismatch!\n");
	else printf("OK!\n");

	free(a);
	free(b);
	free(c);
	free(c2);
	hipFree(a_dev);
	hipFree(b_dev);
	hipFree(c_dev);
	return 0;
}
